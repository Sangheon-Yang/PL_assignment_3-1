/*
 * Find BLANK and replace your own code.
 * And submit report why do you replace the blank that way.
 */


#include <hip/hip_runtime.h>
#include<stdlib.h>
#include<iostream>
#include<fstream>
#include<vector>
#include<string>

#define TILE_WIDTH 6 /* set TILE_WIDTH 16 for the evaluation! */
#define MAXPOOL_INPUT_FILENAME "input.txt"
#define A_FILENAME "a.txt"
#define B_FILENAME "b.txt"
#define C_FILENAME "c.txt"

using namespace std;

__global__ void maxpool(float *input, float *output, const int input_size, const int filter_size) {
    // input : input_matrix address
    // output : output buffer address
    // input_size : width, height of input matrix
    // filter_size : filter_size of maxpolling
    // all input, output matrices are vectorized

    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;

    // out of bound

    // CHANGE

	float tmp = 0.0;
	float Max = -999999.9;
	

	if(col < (input_size/filter_size) && row < (input_size/filter_size)){
	
		for(int i = 0; i < filter_size; i++){
			for(int j = 0; j < filter_size; j++){
				tmp = input[(input_size*filter_size*row)+(filter_size*col)+(input_size*j)+i];	
				if(Max<tmp)
					Max = tmp;
			}
		}	 
		output[((input_size/filter_size)*row)+col] = Max;
	}
	//printf("thread_made\n");
}

__global__ void gemm(float *a, float *b, float *c, const float alpha, const float beta, float *output, const int input_size){
    // a, b, c : input matrix address
    // alpha, beta : input constant
    // output : output buffer address
    // input_size : width, height of input matrix
    // all input, output matrices are vectorized

    int tx = threadIdx.x, ty = threadIdx.y;
    int bx = blockIdx.x,  by = blockIdx.y;

    int row = by*blockDim.y + ty;
    int col = bx*blockDim.x + tx;
    
    if(row>=input_size ||col>=input_size) { return; }
    
    // allocate 2D tiles in __shared__ memory
    __shared__ float s_a[TILE_WIDTH][TILE_WIDTH];
    __shared__ float s_b[TILE_WIDTH][TILE_WIDTH];

    float result = 0;

    // make sure you handle the case when the matrix sizes are not
    // multiple of TILE_WIDTH!
    // loop over the tiles of the input in phases
	
	//1. write one shared memory
	//s_a[ty][tx] = alpha*a[row*input_size + col];
	//s_b[ty][tx] = b[row*input_size + col];
	//__syncthreads();


    for(int p = 0; p < input_size/TILE_WIDTH+1 ;p++){
        // CHANGE

        // You need to use __syncthreads() a few times
        // to synchronize the threads in a thread block.
	
	//2.TILE이 딱 나눠떨어질때
	if(p < input_size/TILE_WIDTH ){
		
		//for(int i = 0 ; i < TILE_WIDTH ; i++)
		//	for(int j= 0; j< TILE_WIDTH; j++)
		s_a[ty][tx] = alpha* a[row*input_size + p*TILE_WIDTH+tx]; 
		//s_b[ty][tx] = b[col*input_size + p*TILE_WIDTH+ty];
		s_b[ty][tx] =b[(ty+p*TILE_WIDTH)*input_size + col];
		__syncthreads();
		for(int i = 0; i<TILE_WIDTH; i++)
			result += s_a[ty][i]*s_b[i][tx];

		//__syncthreads();
	}
	//3.TILE이 딱 안나눠떨어질때
	else if(p == input_size/TILE_WIDTH && input_size%TILE_WIDTH > 0){
		//for(int i = 0 ; i < input_size%TILE_WIDTH ;i++ ){
		//	for(int j = 0; j<input_size%TILE_WIDTH; j++)
		s_a[ty][tx] = alpha* a[row*input_size + p*(TILE_WIDTH)+tx]; 
		//s_b[ty][tx] = b[col*input_size + p*TILE_WIDTH+ty];
		s_b[ty][tx] =b[(ty+p*(TILE_WIDTH))*input_size + col];
		__syncthreads();

		if(blockIdx.y >= input_size/TILE_WIDTH){	
		for(int i = 0; i < TILE_WIDTH; i++){
		//	if(by < input_size/TILE_WIDTH || bx < TILE_WIDTH)
		//		result += s_a[ty][i]*s_b[i][tx];
		//	else if(i < input_size%TILE_WIDTH)
		//		result += s_a[ty][i]*s_b[i][tx];
			result += s_a[ty][i]*s_b[i][tx];
		}
		}
/*
		else if(blockIdx.x == input) {
		for(int i = 0; i < input_size%TILE_WIDTH; i++){
		//	if(by < input_size/TILE_WIDTH || bx < TILE_WIDTH)
		//		result += s_a[ty][i]*s_b[i][tx];
		//	else if(i < input_size%TILE_WIDTH)
		//		result += s_a[ty][i]*s_b[i][tx];
			result += s_a[ty][i]*s_b[i][tx];
		}
		}
*/
		else{
		for(int i = 0; i < input_size%TILE_WIDTH; i++){
		//	if(by < input_size/TILE_WIDTH || bx < TILE_WIDTH)
		//		result += s_a[ty][i]*s_b[i][tx];
		//	else if(i < input_size%TILE_WIDTH)
		//		result += s_a[ty][i]*s_b[i][tx];
			result += s_a[ty][i]*s_b[i][tx];
		}
		}
		
		
	}
	//__syncthreads();
    }
	__syncthreads();
    // write out the result to output[row*input_size + col] 
    // CHANGE
	output[row*input_size + col] = result + (beta * c[row*input_size + col]);
	//__syncthreads(); 
}


int main(int argc, char **argv) {
    if(argc < 4) {
        cout << "usage : " << argv[0] << " input_size filter_size alpha beta\n" << "example : " << argv[0] << " 100 2 0.5 0.8\n";
        return 1;
    }
    const int input_size = stoi(argv[1]);
    const int filter_size = stoi(argv[2]); // used for maxpooling
    const float alpha = stof(argv[3]);
    const float beta = stof(argv[4]);
    const int maxpool_output_size = input_size/filter_size;

    // check input_siize is power of 2
    if(input_size == 0 && (input_size & (input_size-1))){
        cout << "input_size must be power of 2\n";
        return 1;
    }

    if(filter_size == 0){
        cout << "filter_size cannot be 0\n";
        return 1;
    }

    float maxpool_input[input_size*input_size];
    float a[input_size*input_size];
    float b[input_size*input_size];
    float c[input_size*input_size];
    
    // read input matrices 
    ifstream input_in(MAXPOOL_INPUT_FILENAME);
    ifstream a_in(A_FILENAME);
    ifstream b_in(B_FILENAME);
    ifstream c_in(C_FILENAME);

    for (int i = 0; i < input_size*input_size; ++i) {
        input_in >> maxpool_input[i];
        a_in >> a[i];
        b_in >> b[i];
        c_in >> c[i];
    }
    
    // prints inputs for debugging.
    cout<<"filter size : "<<filter_size;
    cout<<"\n========== MAXPOOL_INPUT ==========\n";
    for (int i = 0; i < input_size * input_size; ++i) {
        if(i%input_size==0) cout<<"\n";
        cout<<maxpool_input[i]<<" ";
    }
    cout<<"\nalpha : "<<alpha<<'\n';
    cout<<"========== A ==========\n";
    for (int i = 0; i < input_size * input_size; ++i) {
        if(i%input_size==0) cout<<"\n";
        cout<<a[i]<<" ";
    }
    cout<<"\n========== B ==========\n";
    for (int i = 0; i < input_size * input_size; ++i) {
        if(i%input_size==0) cout<<"\n";
        cout<<b[i]<<" ";
    }
    cout<<"\nbeta : "<<beta<<'\n';
    cout<<"========== C ==========\n";
    for (int i = 0; i < input_size * input_size; ++i) {
        if(i%input_size==0) cout<<"\n";
        cout<<c[i]<<" ";
    }
    cout<<'\n';
       
    // set thread, block dimensions
    const dim3 block_size(TILE_WIDTH, TILE_WIDTH);
    const dim3 num_of_maxpool_blocks(maxpool_output_size/block_size.x+1, maxpool_output_size/block_size.y+1);
    const dim3 num_of_blocks(input_size/block_size.x+1, input_size/block_size.y+1);

    // memory allocation for the device
    float *dev_mem_a, *dev_mem_b, *dev_mem_c, *dev_mem_input, *gemm_output, *maxpool_output;
    hipMalloc(&dev_mem_a, sizeof(float) * input_size * input_size);
    hipMalloc(&dev_mem_b, sizeof(float) * input_size * input_size);
    hipMalloc(&dev_mem_c, sizeof(float) * input_size * input_size);
    hipMalloc(&gemm_output, sizeof(float) * input_size * input_size);
    hipMalloc(&dev_mem_input, sizeof(float) * input_size * input_size);
    hipMalloc(&maxpool_output, sizeof(float) * maxpool_output_size * maxpool_output_size);
    
    // copy variable to device memory
    hipMemcpy(dev_mem_a, a, sizeof(float) * input_size * input_size, hipMemcpyHostToDevice);
    hipMemcpy(dev_mem_b, b, sizeof(float) * input_size * input_size, hipMemcpyHostToDevice);
    hipMemcpy(dev_mem_c, c, sizeof(float) * input_size * input_size, hipMemcpyHostToDevice);
    hipMemcpy(dev_mem_input, maxpool_input, sizeof(float) * input_size * input_size, hipMemcpyHostToDevice);

    // launch CUDA kernels

    // First launch gemm kernel
    gemm<<<num_of_blocks, block_size>>>(dev_mem_a, dev_mem_b, dev_mem_c, alpha, beta, gemm_output, input_size);
    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if(error!=hipSuccess) {
        fprintf(stderr, "ERROR %s\n", hipGetErrorString(error));
        return 1;
    }
 
    // Then run maxpooling
    maxpool<<<num_of_maxpool_blocks, block_size>>>(dev_mem_input, maxpool_output, input_size, filter_size);
    hipDeviceSynchronize();
    error = hipGetLastError();
    if(error!=hipSuccess) {
        fprintf(stderr, "ERROR %s\n", hipGetErrorString(error));
        return 1;
    }
 
    // allocate output buf in main memory
    float *gemm_output_buf = (float*) malloc (sizeof(float)*input_size*input_size);
    float *maxpool_output_buf = (float*) malloc (sizeof(float)*maxpool_output_size*maxpool_output_size);
    
    // copy results from device to host
    hipMemcpy(gemm_output_buf, gemm_output, sizeof(float)*input_size*input_size, hipMemcpyDeviceToHost);
    hipMemcpy(maxpool_output_buf, maxpool_output, sizeof(float)*maxpool_output_size*maxpool_output_size, hipMemcpyDeviceToHost);
    
    // prints the results
    cout<<"\n========== GEMM OUTPUT ==========\n";
    for (int i = 0; i < input_size * input_size; ++i) {
        if(i%input_size==0) cout<<"\n";
        cout<<gemm_output_buf[i]<<" ";
    }
    cout<<"\n========== MAXPOOL OUTPUT ==========\n";
    for (int i = 0; i < maxpool_output_size * maxpool_output_size; ++i) {
        if(i%maxpool_output_size==0) cout<<"\n";
        cout<<maxpool_output_buf[i]<<" ";
    }
    cout<<'\n';

    hipFree(dev_mem_a);
    hipFree(dev_mem_b);
    hipFree(dev_mem_c);
    hipFree(gemm_output);
    hipFree(dev_mem_input);
    hipFree(maxpool_output);
    free(gemm_output_buf);
    free(maxpool_output_buf);
    return 0;
}